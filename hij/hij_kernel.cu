#include "hip/hip_runtime.h"
#include "default.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <torch/extension.h>

__device__ inline int popcnt(unsigned long x) { return __popcll(x); }
__device__ inline int get_parity(unsigned long x) { return __popcll(x) & 1; }
__device__ inline unsigned long get_ones(int n) {
  return (1ULL << n) - 1ULL;
} // parenthesis must be added due to priority
__device__ inline double num_parity(unsigned long x, int i) {
  return (x >> (i - 1) & 1) ? 1.00 : -1.00;
}

__device__ inline int __ctzl(unsigned long x) {
  int r = 63;
  x &= ~x + 1;
  if (x & 0x00000000FFFFFFFF)
    r -= 32;
  if (x & 0x0000FFFF0000FFFF)
    r -= 16;
  if (x & 0x00FF00FF00FF00FF)
    r -= 8;
  if (x & 0x0F0F0F0F0F0F0F0F)
    r -= 4;
  if (x & 0x3333333333333333)
    r -= 2;
  if (x & 0x5555555555555555)
    r -= 1;
  return r;
}

__device__ inline int __clzl(unsigned long x) {
  int r = 0;
  if (!(x & 0xFFFFFFFF00000000))
    r += 32, x <<= 32;
  if (!(x & 0xFFFF000000000000))
    r += 16, x <<= 16;
  if (!(x & 0xFF00000000000000))
    r += 8, x <<= 8;
  if (!(x & 0xF000000000000000))
    r += 4, x <<= 4;
  if (!(x & 0xC000000000000000))
    r += 2, x <<= 2;
  if (!(x & 0x8000000000000000))
    r += 1, x <<= 1;
  return r;
}

__device__ void diff_type(unsigned long *bra, unsigned long *ket, int *p,
                          int _len) {
  unsigned long idiff, icre, iann;
  for (int i = _len - 1; i >= 0; i--) {
    idiff = bra[i] ^ ket[i];
    icre = idiff & bra[i];
    iann = idiff & ket[i];
    p[0] += popcnt(icre);
    p[1] += popcnt(iann);
  }
}

__device__ void get_olst(unsigned long *bra, int *olst, int _len) {
  unsigned long tmp;
  int idx = 0;
  for (int i = 0; i < _len; i++) {
    tmp = bra[i];
    while (tmp != 0) {
      int j = __ctzl(tmp);
      olst[idx] = i * 64 + j;
      tmp &= ~(1ULL << j);
      idx++;
    }
  }
}

__device__ void get_olst(unsigned long *bra, int *olst, int *olst_a,
                         int *olst_b, int _len) {

  unsigned long tmp;
  int ida = 0;
  int idb = 0;
  int idx = 0;
  for (int i = 0; i < _len; i++) {
    tmp = bra[i];
    while (tmp != 0) {
      int j = __ctzl(tmp);
      int s = i * 64 + j;
      olst[idx] = s;
      idx++;
      if (s & 1) {
        olst_b[idb] = s;
        idb++;
      } else {
        olst_a[ida] = s;
        ida++;
      }
      tmp &= ~(1ULL << j);
    }
  }
}

__device__ void get_vlst(unsigned long *bra, int *vlst, int n, int _len) {
  int ic = 0;
  unsigned long tmp;
  for (int i = 0; i < _len; i++) {
    // be careful about the virtual orbital case
    tmp = (i != _len - 1) ? (~bra[i]) : ((~bra[i]) & get_ones(n % 64));
    while (tmp != 0) {
      int j = __ctzl(tmp);
      vlst[ic] = i * 64 + j;
      ic++;
      tmp &= ~(1ULL << j);
    }
  }
}

__device__ void get_vlst(unsigned long *bra, int *vlst, int *vlst_a,
                         int *vlst_b, int n, int _len) {
  int ida = 0;
  int idb = 0;
  int ic = 0;
  unsigned long tmp;
  for (int i = 0; i < _len; i++) {
    // be careful about the virtual orbital case
    tmp = (i != _len - 1) ? (~bra[i]) : ((~bra[i]) & get_ones(n % 64));
    while (tmp != 0) {
      int j = __ctzl(tmp);
      int s = i * 64 + j;
      vlst[ic] = s;
      ic++;
      if (s & 1) {
        vlst_b[idb] = s;
        idb++;
      } else {
        vlst_a[ida] = s;
        ida++;
      }
      tmp &= ~(1ULL << j);
    }
  }
}

__device__ void diff_orb(unsigned long *bra, unsigned long *ket, int _len,
                         int *cre, int *ann) {
  int idx_cre = 0;
  int idx_ann = 0;
  for (int i = _len - 1; i >= 0; i--) {
    unsigned long idiff = bra[i] ^ ket[i];
    unsigned long icre = idiff & bra[i];
    unsigned long iann = idiff & ket[i];
    while (icre != 0) {
      int j = 63 - __clzl(icre); // unsigned long
      cre[idx_cre] = i * 64 + j;
      icre &= ~(1ULL << j);
      idx_cre++;
    }
    while (iann != 0) {
      int j = 63 - __clzl(iann); // unsigned long
      ann[idx_ann] = i * 64 + j;
      iann &= ~(1ULL << j);
      idx_ann++;
    }
  }
}

__device__ int parity(unsigned long *bra, int n) {
  int p = 0;
  for (int i = 0; i < n / 64; i++) {
    p ^= get_parity(bra[i]);
  }
  if (n % 64 != 0) {
    p ^= get_parity((bra[n / 64] & get_ones(n % 64)));
  }
  return -2 * p + 1;
}

__device__ double h1e_get(double *h1e, size_t i, size_t j, size_t sorb) {
  return h1e[j * sorb + i];
}

__device__ double h2e_get(double *h2e, size_t i, size_t j, size_t k, size_t l) {
  if ((i == j) || (k == l))
    return 0.00;
  size_t ij = i > j ? i * (i - 1) / 2 + j : j * (j - 1) / 2 + i;
  size_t kl = k > l ? k * (k - 1) / 2 + l : l * (l - 1) / 2 + k;
  double sgn = 1;
  sgn = i > j ? sgn : -sgn;
  sgn = k > l ? sgn : -sgn;
  double val;
  if (ij >= kl) {
    size_t ijkl = ij * (ij + 1) / 2 + kl;
    val = sgn * h2e[ijkl]; // TODO: value is float64 or tensor ??????
  } else {
    size_t ijkl = kl * (kl + 1) / 2 + ij;
    val = sgn * h2e[ijkl]; // sgn * conjugate(h2e[ijkl])
  }
  return val;
}

__device__ double get_Hii(unsigned long *bra, unsigned long *ket, double *h1e,
                          double *h2e, int sorb, const int nele, int bra_len) {
  double Hii = 0.00;
  int olst[MAX_NELE] = {0};
  get_olst(bra, olst, bra_len);

  for (int i = 0; i < nele; i++) {
    int p = olst[i]; //<p|h|p>
    Hii += h1e_get(h1e, p, p, sorb);
    for (int j = 0; j < i; j++) {
      int q = olst[j];
      Hii += h2e_get(h2e, p, q, p, q); //<pq||pq> Storage not continuous
    }
  }
  return Hii;
}

__device__ double get_HijS(unsigned long *bra, unsigned long *ket, double *h1e,
                           double *h2e, size_t sorb, int bra_len) {
  double Hij = 0.00;
  int p[1], q[1];
  diff_orb(bra, ket, bra_len, p, q);
  Hij += h1e_get(h1e, p[0], q[0], sorb); // hpq
  for (int i = 0; i < bra_len; i++) {
    unsigned long repr = bra[i];
    while (repr != 0) {
      int j = 63 - __clzl(repr);
      int k = 64 * i + j;
      Hij += h2e_get(h2e, p[0], k, q[0], k); //<pk||qk>
      repr &= ~(1ULL << j);
    }
  }
  int sgn = parity(bra, p[0]) * parity(ket, q[0]);
  Hij *= static_cast<double>(sgn);
  return Hij;
}

__device__ double get_HijD(unsigned long *bra, unsigned long *ket, double *h1e,
                           double *h2e, size_t sorb, int bra_len) {
  int p[2], q[2];
  diff_orb(bra, ket, bra_len, p, q);
  int sgn = parity(bra, p[0]) * parity(bra, p[1]) * parity(ket, q[0]) *
            parity(ket, q[1]);
  double Hij = h2e_get(h2e, p[0], p[1], q[0], q[1]);
  Hij *= static_cast<double>(sgn);
  return Hij;
}

/***
__device__ void tensor_to_array(uint8_t *bra_tensor, unsigned long *new_bra, int
len1, int len2)
{
    int idx_bra = 0;
    for(int i=0; i <len2-1; i++){
        unsigned long tmp = 0;
        for(int j=0; j<8; j++){
            unsigned long value = bra_tensor[8*i+j];
            tmp += value << (8*j);
        }
        new_bra[idx_bra] = tmp;
        idx_bra++;
    }
    unsigned long tmp =0;
    for(int i=0; i<len1%8; i++){
        unsigned long value = bra_tensor[(len2-1)*8+i];
        tmp += value << (8*i);
    }
    new_bra[len2-1] =tmp;
}
***/

__device__ double get_Hij(unsigned long *bra, unsigned long *ket, double *h1e,
                          double *h2e, size_t sorb, size_t nele,
                          size_t tensor_len, size_t bra_len) {
  /*
  bra/ket: unsigned long
  */
  double Hij = 0.00;

  int type[2] = {0};
  diff_type(bra, ket, type, bra_len);
  if (type[0] == 0 && type[1] == 0) {
    Hij = get_Hii(bra, ket, h1e, h2e, sorb, nele, bra_len);
  } else if (type[0] == 1 && type[1] == 1) {
    Hij = get_HijS(bra, ket, h1e, h2e, sorb, bra_len);
  } else if (type[0] == 2 && type[1] == 2) {
    Hij = get_HijD(bra, ket, h1e, h2e, sorb, bra_len);
  }
  return Hij;
}

__device__ void get_zvec(unsigned long *bra, double *lst, const int sorb,
                         const int bra_len) {
  int idx = 0;
  for (int i = 0; i < bra_len; i++) {
    for (int j = 1; j <= 64; j++) {
      if (idx >= sorb)
        break;
      lst[idx] = num_parity(bra[i], j);
      idx++;
    }
  }
}

__device__ void get_comb(unsigned long *bra, unsigned long *comb, int n,
                         int len, int noa, int nob, int nva, int nvb) {
  int olst[MAX_NO] = {0};
  int vlst[MAX_NV] = {0};
  int olst_a[MAX_NOA] = {0};
  int olst_b[MAX_NOB] = {0};
  int vlst_a[MAX_NOA] = {0};
  int vlst_b[MAX_NOB] = {0};
  get_olst(bra, olst, olst_a, olst_b, len);
  get_vlst(bra, vlst, vlst_a, vlst_b, n, len);

  for (int i = 0; i < len; i++) {
    comb[i] = bra[i];
  }
  int idx = 1;
  int idx_singles = 0;
  // a->a: noa * nva
  for (int i = 0; i < noa; i++) {
    for (int j = 0; j < nva; j++) {
      int idi = len * idx + olst_a[i] / 64;
      int idj = len * idx + vlst_a[j] / 64;
      comb[idi] = bra[olst_a[i] / 64];
      comb[idj] = bra[vlst_a[j] / 64];
      BIT_FLIP(comb[idi], olst_a[i] % 64);
      BIT_FLIP(comb[idj], vlst_a[j] % 64);
      idx++;
      idx_singles += 1;
    }
  }
  // b->b: nob * nvb
  for (int i = 0; i < nob; i++) {
    for (int j = 0; j < nvb; j++) {
      int idi = len * idx + olst_b[i] / 64;
      int idj = len * idx + vlst_b[j] / 64;
      comb[idi] = bra[olst_b[i] / 64];
      comb[idj] = bra[vlst_b[j] / 64];
      BIT_FLIP(comb[idi], olst_b[i] % 64);
      BIT_FLIP(comb[idj], vlst_b[j] % 64);
      idx++;
      idx_singles++;
    }
  }
  // std::cout << "Singles: " << idx_singles << std::endl;
  int idx_doubles = 0;
  // aa->aa, noa * (noa - 1) * nva * (nva - 1) / 4
  for (int i = 0; i < noa; i++) {
    for (int j = i + 1; j < noa; j++) {
      for (int k = 0; k < nva; k++) {
        for (int l = k + 1; l < nva; l++) {
          int idi = len * idx + olst_a[i] / 64;
          int idj = len * idx + olst_a[j] / 64;
          int idk = len * idx + vlst_a[k] / 64;
          int idl = len * idx + vlst_a[l] / 64;
          comb[idi] = bra[olst_a[i] / 64];
          comb[idj] = bra[olst_a[j] / 64];
          comb[idk] = bra[vlst_a[k] / 64];
          comb[idl] = bra[vlst_a[l] / 64];
          BIT_FLIP(comb[idi], olst_a[i] % 64);
          BIT_FLIP(comb[idj], olst_a[j] % 64);
          BIT_FLIP(comb[idk], vlst_a[k] % 64);
          BIT_FLIP(comb[idl], vlst_a[l] % 64);
          idx++;
          idx_doubles++;
        }
      }
    }
  }
  // bb->bb: nob * (nob - 1) * nvb * (nvb - 1) / 4
  for (int i = 0; i < nob; i++) {
    for (int j = i + 1; j < nob; j++) {
      for (int k = 0; k < nvb; k++) {
        for (int l = k + 1; l < nvb; l++) {
          int idi = len * idx + olst_b[i] / 64;
          int idj = len * idx + olst_b[j] / 64;
          int idk = len * idx + vlst_b[k] / 64;
          int idl = len * idx + vlst_b[l] / 64;
          comb[idi] = bra[olst_b[i] / 64];
          comb[idj] = bra[olst_b[j] / 64];
          comb[idk] = bra[vlst_b[k] / 64];
          comb[idl] = bra[vlst_b[l] / 64];
          BIT_FLIP(comb[idi], olst_b[i] % 64);
          BIT_FLIP(comb[idj], olst_b[j] % 64);
          BIT_FLIP(comb[idk], vlst_b[k] % 64);
          BIT_FLIP(comb[idl], vlst_b[l] % 64);
          idx++;
          idx_doubles++;
        }
      }
    }
  }
  // std::cout << "aa-aa/bb-bb : " << idx_doubles << std::endl;
  // ab->ab (noa * nva * nob * nvb)
  for (int i = 0; i < noa; i++) {
    for (int j = 0; j < nob; j++) {
      for (int k = 0; k < nva; k++) {
        for (int l = 0; l < nvb; l++) {
          int idi = len * idx + olst_a[i] / 64;
          int idj = len * idx + olst_b[j] / 64;
          int idk = len * idx + vlst_a[k] / 64;
          int idl = len * idx + vlst_b[l] / 64;
          comb[idi] = bra[olst_a[i] / 64];
          comb[idj] = bra[olst_b[j] / 64];
          comb[idk] = bra[vlst_a[k] / 64];
          comb[idl] = bra[vlst_b[l] / 64];
          BIT_FLIP(comb[idi], olst_a[i] % 64);
          BIT_FLIP(comb[idj], olst_b[j] % 64);
          BIT_FLIP(comb[idk], vlst_a[k] % 64);
          BIT_FLIP(comb[idl], vlst_b[l] % 64);
          idx++;
          idx_doubles++;
        }
      }
    }
  }
}

__global__ void get_zvec_kernel_3D(double *comb_ptr, unsigned long *bra,
                                   const size_t sorb, const size_t bra_len,
                                   int n, int m) {
  int idn = blockIdx.x * blockDim.x + threadIdx.x;
  int idm = blockIdx.y * blockDim.y + threadIdx.y;
  if (idn >= n || idm >= m)
    return;
  get_zvec(&bra[idn * m * bra_len + idm * bra_len],
           &comb_ptr[idn * m * sorb + idm * sorb], sorb, bra_len);
}

__global__ void get_zvec_kernel_2D(double *comb_ptr, unsigned long *bra,
                                   const size_t sorb, const size_t bra_len,
                                   int n) {
  int idn = blockIdx.x;
  int idm = blockIdx.x * blockDim.x + threadIdx.x;
  if (idm >= n)
    return;
  // printf("idn/idm : %d/%d \n", idn, idm);
  get_zvec(&bra[idm], &comb_ptr[idm * sorb], sorb, bra_len);
}

__global__ void get_Hij_kernel_2D(double *Hmat_ptr, unsigned long *bra,
                                  unsigned long *ket, double *h1e, double *h2e,
                                  const size_t sorb, const size_t nele,
                                  const size_t tensor_len, const size_t bra_len,
                                  int n, int m) {
  int idn = blockIdx.x * blockDim.x + threadIdx.x;
  int idm = blockIdx.y * blockDim.y + threadIdx.y;
  if (idn >= n || idm >= m)
    return;

  Hmat_ptr[idn * m + idm] = get_Hij(&bra[idn * bra_len], &ket[idm * bra_len],
                                    h1e, h2e, sorb, nele, tensor_len, bra_len);
}

__global__ void get_Hij_kernel_3D(double *Hmat_ptr, unsigned long *bra,
                                  unsigned long *ket, double *h1e, double *h2e,
                                  const size_t sorb, const size_t nele,
                                  const size_t tensor_len, const size_t bra_len,
                                  int n, int m) {
  int idn = blockIdx.x * blockDim.x + threadIdx.x;
  int idm = blockIdx.y * blockDim.y + threadIdx.y;
  if (idn >= n || idm >= m)
    return;
  Hmat_ptr[idn * m + idm] =
      get_Hij(&bra[idn * bra_len], &ket[idn * m * bra_len + idm * bra_len], h1e,
              h2e, sorb, nele, tensor_len, bra_len);
}

__global__ void get_comb_kernel_2D(unsigned long *bra_ptr,
                                   unsigned long *comb_ptr, int sorb, int len,
                                   int noa, int nob, int nva, int nvb,
                                   int nbatch, int ncomb) {
  int idn = blockIdx.x;
  int idm = blockIdx.x * blockDim.x + threadIdx.x;
  if (idm >= nbatch)
    return;
  // comb_ptr [nbatch, ncomb, sorb]
  // printf("idn/idm : %d/%d  %d\n", idn, idm, nbatch);
  get_comb(&bra_ptr[idm], &comb_ptr[idm * ncomb * len], sorb, len, noa, nob, nva,
           nvb);
}

torch::Tensor get_Hij_cuda(torch::Tensor &bra_tensor, torch::Tensor &ket_tensor,
                           torch::Tensor &h1e_tensor, torch::Tensor &h2e_tensor,
                           const int sorb, const int nele) {
  /*
  bra_tensor: shape(N, a): a =
  ket_tensor: shape(M, a): a = ((sorb-1)/64 + 1)
  h1e_tensor/h2e_tensor: one dim
  sorb: the number of spin orbital
  nele: the number of eletron
  */

  // GPU time: https://www.jianshu.com/p/424db3a33ca9
  hipEvent_t t0, t1;
  hipEventCreate(&t0);
  hipEventCreate(&t1);
  hipEventRecord(t0);

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  const int ket_dim = ket_tensor.dim();
  if (VERBOSE) {
    std::cout << "ket dim: " << ket_dim << std::endl;
  }
  bool flag_3d = false;
  const int tensor_len = (sorb - 1) / 8 + 1;
  const int bra_len = (sorb - 1) / 64 + 1;
  int n, m;
  if (ket_dim == 3) {
    flag_3d = true;
    // bra: (n, tensor_len), ket: (n, m, tensor_len)
    n = bra_tensor.size(0), m = ket_tensor.size(1);
  } else if (ket_dim == 2) {
    flag_3d = false;
    // bra: (n, tensor_len), ket: (m, tensor_len)
    n = bra_tensor.size(0), m = ket_tensor.size(0);
  } else {
    // do not throw exception
    throw "ket dim error";
  }

  torch::Tensor Hmat = torch::zeros({n, m}, h1e_tensor.options());
  hipDeviceSynchronize();

  double *h1e_ptr = h1e_tensor.data_ptr<double>();
  double *h2e_ptr = h2e_tensor.data_ptr<double>();
  unsigned long *bra_ptr =
      reinterpret_cast<unsigned long *>(bra_tensor.data_ptr<uint8_t>());
  unsigned long *ket_ptr =
      reinterpret_cast<unsigned long *>(ket_tensor.data_ptr<uint8_t>());
  double *Hmat_ptr = Hmat.data_ptr<double>();

  dim3 threads(THREAD, THREAD);
  dim3 blocks((n + threads.x - 1) / threads.x, (m + threads.y - 1) / threads.y);

  // std::cout << "threads: " << THREAD << " " << THREAD << std::endl;
  hipEventRecord(end);
  hipEventSynchronize(end);
  float time_ms = 0.f;
  hipEventElapsedTime(&time_ms, start, end);
  if (VERBOSE) {
    std::cout << std::setprecision(6);
    std::cout << "GPU Hmat initialization time: " << time_ms << " ms"
              << std::endl;
  }

  hipEvent_t start0, end0;
  hipEventCreate(&start0);
  hipEventCreate(&end0);
  hipEventRecord(start0);
  if (flag_3d) {
    get_Hij_kernel_3D<<<blocks, threads>>>(Hmat_ptr, bra_ptr, ket_ptr, h1e_ptr,
                                           h2e_ptr, sorb, nele, tensor_len,
                                           bra_len, n, m);
  } else {
    get_Hij_kernel_2D<<<blocks, threads>>>(Hmat_ptr, bra_ptr, ket_ptr, h1e_ptr,
                                           h2e_ptr, sorb, nele, tensor_len,
                                           bra_len, n, m);
  }
  hipDeviceSynchronize();
  hipEventRecord(end0);
  hipEventSynchronize(end0);
  float kernel_time_ms = 0.f;
  hipEventElapsedTime(&kernel_time_ms, start0, end0);
  if (VERBOSE) {
    std::cout << std::setprecision(6);
    std::cout << "GPU calculate <n|H|m> time: " << kernel_time_ms << " ms"
              << std::endl;
  }

  hipEventRecord(t1);
  hipEventSynchronize(t1);
  float total_time_ms = 0.f;
  hipEventElapsedTime(&total_time_ms, t0, t1);
  if (VERBOSE) {
    std::cout << std::setprecision(6);
    std::cout << "Total function GPU function time: " << total_time_ms
              << " ms\n"
              << std::endl;
  }

  return Hmat;
}

torch::Tensor get_comb_tensor_cuda(torch::Tensor &bra_tensor, const int sorb,
                                   const int nele, bool ms_equal) {
  // TODO: how to accelerate get_comb funciton??? 
  const int no = nele;
  const int nv = sorb - nele;
  const int bra_len = (sorb - 1) / 64 + 1;
  const int nob = nele / 2, noa = no - nob;
  const int nvb = nv / 2, nva = nv - nvb;
  int nsingles, ndoubles, ncomb;
  ms_equal = true;
  if (ms_equal) {
    nsingles = noa * nva + nob * nvb;
    ndoubles = noa * (noa - 1) * nva * (nva - 1) / 4 +
               nob * (nob - 1) * nvb * (nvb - 1) / 4 + noa * nva * nob * nvb;
  } else {
    // TODO: ms is not equal, how to achieve??
    nsingles = no * nv;
    ndoubles = no * (no - 1) * nv * (nv - 1) / 4;
  }
  ncomb = 1 + nsingles + ndoubles;
  // bra_tensor(batch, sorb) or (sorb)
  const int nbatch = bra_tensor.size(0);
  const int dim = bra_tensor.dim();
  bool flag_3d = false;
  torch::Tensor comb;
  auto options = bra_tensor.options();
  if ((dim == 1) or (nbatch == 1 && dim == 2)) {
    comb = torch::zeros({ncomb, 8 * bra_len}, options);
  } else if (nbatch > 1 && dim == 2) {
    flag_3d = true;
    comb = torch::zeros({nbatch, ncomb, 8 * bra_len}, options);
  } else {
    std::cout << "bra shape maybe error:" << bra_tensor.sizes() << std::endl;
    throw "0";
  }
  unsigned long *bra_ptr =
      reinterpret_cast<unsigned long *>(bra_tensor.data_ptr<uint8_t>());
  unsigned long *comb_ptr =
      reinterpret_cast<unsigned long *>(comb.data_ptr<uint8_t>());

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  if (flag_3d) {
    dim3 threads(1024);
    dim3 blocks((nbatch + threads.x - 1) / threads.x);
    get_comb_kernel_2D<<<blocks, threads>>>(bra_ptr, comb_ptr, sorb, bra_len,
                                            noa, nob, nva, nvb, nbatch, ncomb);
  } else {
    std::cout << "Do not know how to achieve" << std::endl;
    throw "0";
    // get_comb_kernel_2D<<<1, 1>>>(bra_ptr, comb_ptr, sorb, bra_len, noa, nob,
    // nva, nvb, 1, comb);
  }
  hipDeviceSynchronize();
  hipEventRecord(end);
  hipEventSynchronize(end);
  float kernel_time_ms = 0.f;
  hipEventElapsedTime(&kernel_time_ms, start, end);
  if (VERBOSE) {
    std::cout << std::setprecision(6);
    std::cout << "GPU calculate (Singles and Doubls combination) time: "
              << kernel_time_ms << "ms\n"
              << std::endl;
  }
  return comb;
}

torch::Tensor uint8_to_bit_cuda(torch::Tensor &bra_tensor, const int sorb) {
  bool flag_3d;
  const int bra_len = (sorb - 1) / 64 + 1;
  const int bra_dim = bra_tensor.dim();
  int n = 0, m = 0;
  torch::Tensor comb_bit;
  auto options = torch::TensorOptions()
                     .dtype(torch::kDouble)
                     .layout(bra_tensor.layout())
                     .device(bra_tensor.device())
                     .requires_grad(false);

  if (bra_dim == 3) {
    flag_3d = true;
    // [batch, ncomb, sorb]
    n = bra_tensor.size(0), m = bra_tensor.size(1);
    comb_bit = torch::zeros({n, m, sorb}, options);
    // dim3 threads(THREAD, THREAD);
    // dim3 blocks((n+threads.x-1)/threads.x, (m+threads.y-1)/threads.y);
  } else if (bra_dim == 2) {
    flag_3d = false;
    // [ncomb, sorb]
    n = bra_tensor.size(0);
    comb_bit = torch::zeros({n, sorb}, options);
    // dim3 threads = (512);
    // dim3 blocks((n+threads.x-1)/threads.x);
  } else {
    throw "bra dim error";
  }

  unsigned long *bra_ptr =
      reinterpret_cast<unsigned long *>(bra_tensor.data_ptr<uint8_t>());
  double *comb_ptr = comb_bit.data_ptr<double>();

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  if (flag_3d) {
    dim3 threads(THREAD, THREAD);
    dim3 blocks((n + threads.x - 1) / threads.x,
                (m + threads.y - 1) / threads.y);
    get_zvec_kernel_3D<<<blocks, threads>>>(comb_ptr, bra_ptr, sorb, bra_len, n,
                                            m);
  } else {
    dim3 threads(1024);
    dim3 blocks((n + threads.x - 1) / threads.x);
    get_zvec_kernel_2D<<<blocks, threads>>>(comb_ptr, bra_ptr, sorb, bra_len,
                                            n);
  }
  hipDeviceSynchronize();
  hipEventRecord(end);
  hipEventSynchronize(end);
  float kernel_time_ms = 0.f;
  hipEventElapsedTime(&kernel_time_ms, start, end);
  if (VERBOSE) {
    std::cout << std::setprecision(6);
    std::cout << "GPU calculate comb(unit8->bit) time: " << kernel_time_ms
              << "ms\n"
              << std::endl;
  }

  return comb_bit;
}
